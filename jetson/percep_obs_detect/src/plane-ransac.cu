#include "hip/hip_runtime.h"
#include "plane-ransac.hpp"
#include "filter.hpp"
#include <stdlib.h>
#include <unistd.h>
#include <thrust/extrema.h>


#define DEBUG

// TODO: move this into common
__device__ int ceilDivGPU(int a, int b) {
    return (a + b - 1) / b;
}

/* 
LAUNCH:
    - [Block] # iterations [aka, randomly selected models] to try
    - [Thread] MAX_THREADS

REQUIRES:
    - GPU point cloud
    - A buffer to write inlier counts for each attempted model
    - A buffer that tells the kernel what the randomly selected points were for each model
    - Threshold distance for a pt to be considered an inlier
EFFECTS:

    [Block]: 
    Each block represents an "iteration" of the traditional RANSAC algorithm. 
    That is, every block has a different set of randomly chosen 3 points that define the
    model (a plane is minimally defined by 3 points). The threads in the block then check
    every point in the point cloud against the model for that block.

    [Thread]:
    Threads are used to decide how many points are inliers to the model. If
    thread max = 1024 and there are 2048 points, each thread will process 2
    points. Each thread will write the number of inliers in the set of points it evaluated
    to its specific spot in shared memory. Threads are synced, and then threads will 
    participate in a parallel reduction to give the total number of inliers for that block/model, 
    which will be returned from the kernel in the inlierCounts buffer. 
*/
__global__ void ransacKernel(GPU_Cloud pc, float* inlierCounts, int* modelPoints, float threshold, float3 axis, float epsilon) { 
    __shared__ float inlierField[MAX_THREADS];
    inlierField[threadIdx.x] = 0;

    int iteration = blockIdx.x; //which "iteration" of RANSAC
    float inliers = 0; //number of inliers in this thread

    // select 3 random points from the cloud as the model that this particular block will evaluate
    int randIdx0 = modelPoints[iteration*3 + 0];
    int randIdx1 = modelPoints[iteration*3 + 1];
    int randIdx2 = modelPoints[iteration*3 + 2];

    if(randIdx0 >= pc.size || randIdx1 >= pc.size || randIdx2 >= pc.size) {
        inlierCounts[iteration] = 0;
        return;
    }

    float3 modelPt0 = make_float3(pc.data[randIdx0].x, pc.data[randIdx0].y, pc.data[randIdx0].z);
    float3 modelPt1 = make_float3(pc.data[randIdx1].x, pc.data[randIdx1].y, pc.data[randIdx1].z);
    float3 modelPt2 = make_float3(pc.data[randIdx2].x, pc.data[randIdx2].y, pc.data[randIdx2].z);    

    // Create a plane from the 3 points
    Plane plane(modelPt0, modelPt1, modelPt2);

    //check that n dot desired axis is less than epsilon, if so, return here 
    if(abs(dot(normalize(plane.normal), normalize(axis))) < epsilon) {
        if(threadIdx.x == 0) inlierCounts[iteration] = 0; //make it -1 to show invalid model?
        return;
    }

    // Construct predicate to chek if a point is an inlier in the plane
    NotInPlane pred(plane.normal, modelPt1, threshold);

    // figure out how many points each thread must compute distance for and determine if each is inlier/outlier
    int pointsPerThread = ceilDivGPU(pc.size, MAX_THREADS);
    for(int i = 0; i < pointsPerThread; i++) {
        // select a point index or return if this isn't a valid point
        int pointIdx = threadIdx.x * pointsPerThread + i;
        if(pointIdx >= pc.size) continue; //TODO Should this be return??? 
        
        // point in the point cloud that could be an inlier or outlier
        float4 curPt = make_float4(pc.data[pointIdx].x, pc.data[pointIdx].y, pc.data[pointIdx].z, 0);
        
        //add a 1 if inlier in plane, 0 if not 
        inliers += (pred(curPt)) ? 0 : 1; //very probalmatic line, how can we reduce these checks
    }
    
    //parallel reduction to get an aggregate sum of the number of inliers for this model
    //this is all equivalent to sum(inlierField), but it does it in parallel
    inlierField[threadIdx.x] = inliers;
    __syncthreads();
    int aliveThreads = (blockDim.x) / 2;
	while (aliveThreads > 0) {
		if (threadIdx.x < aliveThreads) {
            inliers += inlierField[aliveThreads + threadIdx.x];
			if (threadIdx.x >= (aliveThreads) / 2) inlierField[threadIdx.x] = inliers;
		}
		__syncthreads();
		aliveThreads /= 2;
	}

    //at the final thread, write to global memory
    if(threadIdx.x == 0) {
        inlierCounts[iteration] = inliers;
    } 
}

 /**
  * \brief Updates the plane selection from the cloud using the given model index 
  */
__global__ void getOptimalModelPoints(GPU_Cloud pc, Plane &selection, int idx, int* modelPoints, float* maxCount) {
    int pt = threadIdx.x;
    float4 point = pc.data[modelPoints[3*idx + pt]];
    selection[pt] = make_float3(point.x, point.y, point.z);

    // Use one thread to compute the normal
    __syncthreads();
    if(threadIdx.x == 0) {
        selection.ComputeNormal();

        #ifdef DEBUG
        printf("Winner model inlier count: %f \n", *maxCount);
        #endif
    }
}

void RansacPlane::selectOptimalModel() {
    float* maxCount = thrust::max_element(thrust::device, inlierCounts, inlierCounts + iterations);
    // Pointer arithmetic gives us the model index with most inliers
    int maxIdx = maxCount - inlierCounts;
    // Send the index to GPU
    hipMemcpy(optimalModelIndex, &maxIdx , sizeof(int), hipMemcpyHostToDevice);
    // Now launch a kernel to write the Plane of this model into selection
    getOptimalModelPoints<<<1, 3>>>(pc, *selection, maxIdx, modelPoints, maxCount);
    checkStatus(hipDeviceSynchronize());
}

RansacPlane::RansacPlane(float3 axis, float epsilon, int iterations, float threshold, int pcSize, float removalRadius)
    : pc(pc), axis(axis), epsilon(epsilon), iterations(iterations), threshold(threshold), removalRadius(removalRadius)  {
    
    //Set up buffers needed for RANSAC
    hipMalloc(&inlierCounts, sizeof(float) * iterations); 
    hipMalloc(&modelPoints, sizeof(int) * iterations * 3);
    hipMalloc(&selection, sizeof(Plane));
    hipMalloc(&optimalModelIndex, sizeof(int));

    //Generate random numbers in CPU to use in RANSAC kernel
    int* randomNumsCPU = (int*) malloc(sizeof(int) * iterations* 3);

    for(int i = 0; i < iterations; i++) {
        int a = 0;
        int b = 0;
        int c = 0;
        while(a == b || b == c || a == c) {
            a = rand() % pcSize;
            b = rand() % pcSize;
            c = rand() % pcSize;
        }
    
        randomNumsCPU[i*3] = a;
        randomNumsCPU[i*3 + 1] = b;
        randomNumsCPU[i*3 + 2] = c; 
    }

    hipMemcpy(modelPoints, randomNumsCPU, sizeof(int) * iterations * 3, hipMemcpyHostToDevice);
    free(randomNumsCPU);

    // Generate a buffer for retreiving the selected model from CUDA Kernels
    selectedModel = (Plane*) malloc(sizeof(Plane)); 
}

Plane RansacPlane::computeModel(GPU_Cloud &pc) {
    if(pc.size == 0) return {make_float3(0,0,0), make_float3(0,0,0), make_float3(0,0,0)};

    // Copy vars locally
    this->pc = pc;
    int blocks = iterations;
    int threads = MAX_THREADS;
    
    // Get a list of models and corresponding inlier count
    ransacKernel<<<blocks, threads>>>(pc, inlierCounts, modelPoints, threshold, axis, cos(epsilon*3.1415/180));
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());

    // Choose the model with the greatest inlier count
    selectOptimalModel();

    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());   

    // Copy selected plane to CPU
    hipMemcpy(selectedModel, selection, sizeof(Plane), hipMemcpyDeviceToHost);
    
    // Filter out all the points in the plane
    NotInPlane predicate(selectedModel->normal, selectedModel->p1, threshold);
    Filter<NotInPlane>(pc, predicate, FilterOp::REMOVE, 0);
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());

    return *selectedModel;
}

RansacPlane::~RansacPlane() {
    hipFree(inlierCounts);
    hipFree(modelPoints);
    hipFree(selection);
    hipFree(optimalModelIndex);
    free(selectedModel);
}