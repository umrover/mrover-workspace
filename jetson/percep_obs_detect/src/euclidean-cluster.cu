#include "hip/hip_runtime.h"
#include "euclidean-cluster.hpp"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include "common.hpp"
#include <limits>     //for std::numeric_limits<> 

//Helper functions
__device__ float getFloatData(int axis, float4 &val) {
    if(!axis)
        return val.x;
    else if(axis == 1)
        return val.y;
    else
        return val.z;
}
            
__device__ float getData(int axis, int index, float4 *data) {
    return getFloatData(axis, data[index]); 
}

__global__ void determineGraphStructureKernelN2(GPU_Cloud pc, float tolerance, int* listStart) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;
    if (ptIdx == 0) printf("OK\n");

    float4 ptLocal = pc.data[ptIdx];
    float3 pt = make_float3(ptLocal.x, ptLocal.y, ptLocal.z);
    int neighborCount = 0;
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {
        float4 tmpPtLocal = pc.data[i];
        float3 dvec = (pt - make_float3(tmpPtLocal.x, tmpPtLocal.y, tmpPtLocal.z));
        //this is a neighbor
        if( length(dvec) < tolerance && i != ptIdx) {
            neighborCount++;
        }
    }
    listStart[ptIdx] = neighborCount;

    //we must do an exclusive scan using thrust after this kernel
    if (ptIdx == 0) printf("Nice\n");
}


/* This kernel builds the graph 
Fairly standard adjacency list structure. 
*/
__global__ void buildGraphKernelN2(GPU_Cloud pc, float tolerance, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    float4 ptLocal = pc.data[ptIdx];
    float3 pt = make_float3(ptLocal.x, ptLocal.y, ptLocal.z);
    int neighborCount = 0;
    //get the adjacency list for this point
    int* list = neighborLists + listStart[ptIdx];
    
    //horrible slow way of doing this that is TEMPORARY --> please switch to radix sorted bins
    for(int i = 0; i < pc.size; i++) {

        float4 tmpPtLocal = pc.data[i];
        float3 dvec = (pt - make_float3(tmpPtLocal.x, tmpPtLocal.y, tmpPtLocal.z));
        //this is a neighbor
        if( length(dvec) < tolerance && i != ptIdx) {
            list[neighborCount] = i;
            neighborCount++;
        }
    }
    
    labels[ptIdx] = ptIdx;
    f1[ptIdx] = true;
    f2[ptIdx] = false;
}

__device__ int numNeighborsForBin(GPU_Cloud pc, float tolerance, Bins bins, int ptIdx, int partitions, float3 pt, int binNum) {
    if ((binNum < 0) || (binNum >= partitions * partitions * partitions)) return 0;
    int neighborCount = 0;
    for(int j = 0; j < bins.data[binNum + 1] - bins.data[binNum]; ++j) {
        // Iterates through points in the bin
        float4 pcPointData = pc.data[j + bins.data[binNum]];
        float3 dvec = (pt - make_float3(pcPointData.x, pcPointData.y, pcPointData.z));
        if (length(dvec) < tolerance && j + bins.data[binNum] != ptIdx) ++neighborCount;
    }
    return neighborCount;
}

__global__ void determineGraphStructureKernel(GPU_Cloud pc, float tolerance, int* listStart, Bins bins) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    float3 pt = make_float3(pc.data[ptIdx].x, pc.data[ptIdx].y, pc.data[ptIdx].z);
    int neighborCount = 0;
    int currBinNum = pc.data[ptIdx].w;
    int partitions = bins.partition;
    int partitionsSquared = partitions * partitions;

    int currBinX = currBinNum / partitionsSquared;
    int currBinY = (currBinNum % partitionsSquared) / partitions;
    int currBinZ = currBinNum % partitions;

    /*
    The bin one away from currBin in Z direction is binNum + 1
    The bin one away from currBin in Y direction is binNum + partitions
    The bin one away from currBin in X direction is binNum + partitions^2
    */
    // Check the current bin
    neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum);
    // Check bin with coordinate x + 1
    if (currBinX != (partitions - 1))
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum + partitionsSquared);
    // Check bin with coordinate y + 1
    if (currBinY != (partitions - 1))
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum + partitions);
    // Check bin with coordinate z + 1
    if (currBinZ != (partitions - 1))
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum + 1);
    // Check bin with coordinate x - 1
    if (currBinX != 0)
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum - partitionsSquared);
    // Check bin with coordinate y - 1
    if (currBinY != 0)
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum - partitions);
    // Check bin with coordinate z - 1
    if (currBinZ != 0)
        neighborCount += numNeighborsForBin(pc, tolerance, bins, ptIdx, partitions, pt, currBinNum - 1);
    listStart[ptIdx] = neighborCount;

}


__device__ int populateNeighborList(GPU_Cloud pc, float tolerance, Bins bins, int ptIdx, int partitions, float3 pt, int* list, int binNum, int count) {
    if ((binNum < 0) || (binNum >= partitions * partitions * partitions)) return;
    for(int j = 0; j < bins.data[binNum + 1] - bins.data[binNum]; ++j) {
        // Iterates through points in the bin
        float4 pcPointData = pc.data[j + bins.data[binNum]];
        float3 dvec = (pt - make_float3(pcPointData.x, pcPointData.y, pcPointData.z));
        if (length(dvec) < tolerance && j + bins.data[binNum] != ptIdx) {
            list[count] = j + bins.data[binNum];
            count++;
        }
    }
    return count;
}

/* This kernel builds the graph
Fairly standard adjacency list structure. 
*/
__global__ void buildGraphKernel(GPU_Cloud pc, float tolerance, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2, Bins bins) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    float3 pt = make_float3(pc.data[ptIdx].x, pc.data[ptIdx].y, pc.data[ptIdx].z);
    int neighborCount = 0;
    int currBinNum = pc.data[ptIdx].w;
    int partitions = bins.partition;
    int partitionsSquared = partitions * partitions;

    int currBinX = currBinNum / partitionsSquared;
    int currBinY = (currBinNum % partitionsSquared) / partitions;
    int currBinZ = currBinNum % partitions;

    int* list = neighborLists + listStart[ptIdx];

    /*
    The bin one away from currBin in Z direction is binNum + 1
    The bin one away from currBin in Y direction is binNum + partitions
    The bin one away from currBin in X direction is binNum + partitions^2
    */
    // Check the current bin
    neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum, neighborCount);
    // Check bin with coordinate x + 1
    if (currBinX != (partitions - 1))
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum + partitionsSquared, neighborCount);
    // Check bin with coordinate y + 1
    if (currBinY != (partitions - 1))
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum + partitions, neighborCount);
    // Check bin with coordinate z + 1
    if (currBinZ != (partitions - 1))
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum + 1, neighborCount);
    // Check bin with coordinate x - 1
    if (currBinX != 0)
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum - partitionsSquared, neighborCount);
    // Check bin with coordinate y - 1
    if (currBinY != 0)
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum - partitions, neighborCount);
    // Check bin with coordinate z - 1
    if (currBinZ != 0)
        neighborCount = populateNeighborList(pc, tolerance, bins, ptIdx, partitions, pt, list, currBinNum - 1, neighborCount);

    labels[ptIdx] = ptIdx;
    f1[ptIdx] = true;
    f2[ptIdx] = false;
}

/*
this kernel propogates labels, it must be called in a loop until its flag "m" is false, indicating
no more changes are pending. 
*/
//each thread is a point 
__global__ void propogateLabels(GPU_Cloud pc, int* neighborLists, int* listStart, int* labels, bool* f1, bool* f2, bool* m) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    if(ptIdx == -1){
        for(int i = 0; i < 10; i++){
            printf("Pt %i: ", i);
            for(int j = listStart[i]; j < listStart[i+1]; ++j){
                printf("%i, ", neighborLists[j]);
            }
            printf("\n");    
        }
        
    }
    //debug lines
   // if(threadIdx.x == 0) *m = false;
   // __syncthreads();
   // printf("pt idx: %d, label: %d, flag: %d frontier one: %d frontier two: %d \n", ptIdx, labels[ptIdx], (*m) ? 1 : 0, f1[ptIdx] ? 1 : 0, f2[ptIdx] ? 1 : 0);

    bool mod = false;
    //TODO, load the NEIGHBOR list to shared memory 
    if(f1[ptIdx]) {
        //printf("active frontier %d \n", ptIdx);

        int* list = neighborLists + listStart[ptIdx];
        int listLen = listStart[ptIdx+1] - listStart[ptIdx];
        f1[ptIdx] = false;
        int myLabel = labels[ptIdx];

        //printf("[len] pt idx: %d, list-len: %d \n", ptIdx, listLen);

        for(int i = 0; i < listLen; i++) {
            int otherLabel = labels[list[i]];
            if(myLabel < otherLabel) { //are these reads actually safe?
                //printf("-- updating other: %d to be %d \n", otherLabel, myLabel);

                atomicMin(&labels[list[i]], myLabel);
                f2[list[i]] = true;
                *m = true;
            } else if(myLabel > otherLabel) {
                myLabel = otherLabel;
                mod = true;
            }
        }

        if(mod) {
            atomicMin(&labels[ptIdx], myLabel);
            f2[ptIdx] = true;
            *m = true;
        }
    } 

    /*
    __syncthreads();
    if(threadIdx.x == 0) {
    if(*m) printf("still going \n");
    else printf("done \n");
    }*/
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

    return old;
}


__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

//this debug kernel colors points based on their label
__global__ void colorClusters(GPU_Cloud pc, int* labels, int* keys, int* values, int minCloudSize, int numClusters, float* minX, float* maxX, float* minY, float* maxY, float* minZ, float* maxZ) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    //DEBUG STEP REMOVE
    //pc.data[ptIdx].w = 9.18340948595e-41;
    //return;

    int i = 0;
    while(true) {
        if(labels[ptIdx] == keys[i]) {
            if(values[i] < minCloudSize) {
                pc.data[ptIdx].w = VIEWER_BGR_COLOR;
                return;
            }
            else break;
        }
        i++;
    }
    
    //float red = 3.57331108403e-43;
    //float green = 9.14767637511e-41;
    //float blue = 2.34180515203e-38;
    //float magenta = 2.34184088514e-38; 
    float yellow = 9.18340948595e-41;
    
    pc.data[ptIdx].w = yellow+0.0000000000000001*labels[ptIdx]*4;
    
    //X
    atomicMinFloat(&minX[i], pc.data[ptIdx].x);
    atomicMaxFloat(&maxX[i], pc.data[ptIdx].x);

    //Y
    atomicMinFloat(&minY[i], pc.data[ptIdx].y);
    atomicMaxFloat(&maxY[i], pc.data[ptIdx].y);

    //Z
    atomicMinFloat(&minZ[i], pc.data[ptIdx].z);
    atomicMaxFloat(&maxZ[i], pc.data[ptIdx].z);
}

//this is practically serial, can we just color using OpenGL functions
__global__ void colorExtrema(GPU_Cloud pc, int* values, int minSize, int* labels, int numClustersOrig, int* validClustersCount, float* minX, float* maxX,  float* minY, float* maxY, float* minZ, float* maxZ) {
    int clusterIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(clusterIdx >= numClustersOrig) return;

    int place = 0;
    if(values[clusterIdx] > minSize) place = atomicAdd(validClustersCount, 1);
    else return;

  
    pc.data[place*2] = make_float4(minX[clusterIdx], (minY[clusterIdx] + maxY[clusterIdx])/2, minZ[clusterIdx], 0.0);
    pc.data[place*2+1] = make_float4(maxX[clusterIdx], (minY[clusterIdx] + maxY[clusterIdx])/2, minZ[clusterIdx], 0.0);
        
    //serailze the extrema into a float4 vector using the "place"
}

__global__ void colorClustersNew(GPU_Cloud pc, int* labels, int* keys, int numClusters) {
    int ptIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(ptIdx >= pc.size) return;

    float yellow = 9.18340948595e-41;

    for(int i = 0; i < numClusters; i++) {
        if(labels[ptIdx] == keys[i]) {
            pc.data[ptIdx].w = yellow+0.0000000000000001*labels[ptIdx]*4;
            return;
        }
    }

    pc.data[ptIdx].w = VIEWER_BGR_COLOR;

}

    
class is_smaller_than_min {
public: 
    is_smaller_than_min(int min) : min(min) {}
    __device__ __host__ bool operator()(const int size) {
        return size < min;
    }
private:
    int min;
};

EuclideanClusterExtractor::EuclideanClusterExtractor(float tolerance, int minSize, float maxSize, size_t cloudArea, int partitions) 
: tolerance{tolerance}, minSize{minSize}, maxSize{maxSize}, partitions{partitions} {

    hipMalloc(&listStart, sizeof(int)*(cloudArea+1));
    hipMalloc(&labels, sizeof(int)*cloudArea);
    hipMalloc(&f1, sizeof(bool)*cloudArea);
    hipMalloc(&f2, sizeof(bool)*cloudArea);
    hipMalloc(&stillGoing, sizeof(bool));

}

EuclideanClusterExtractor::EuclideanClusterExtractor() {}

//perhaps use dynamic parallelism 
EuclideanClusterExtractor::ObsReturn EuclideanClusterExtractor::extractClusters(GPU_Cloud &pc, Bins &bins) {
    ObsReturn empty;
    empty.size = 0;
    if(pc.size == 0) return empty;
    // Find the structure for adjacency list of all points
    #if !VOXEL
        determineGraphStructureKernelN2<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, listStart);
        checkStatus(hipGetLastError());
        checkStatus(hipDeviceSynchronize());
    #endif
    #if VOXEL
       determineGraphStructureKernel<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, listStart, bins);
       checkStatus(hipGetLastError());
       checkStatus(hipDeviceSynchronize());
    #endif
    thrust::exclusive_scan(thrust::device, listStart, listStart+pc.size+1, listStart, 0);
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());
    
    // Create helpful variables
    int totalAdjanecyListsSize;
    checkStatus(hipMemcpy(&totalAdjanecyListsSize, &listStart[pc.size], sizeof(int), hipMemcpyDeviceToHost));
    hipMalloc(&neighborLists, sizeof(int)*totalAdjanecyListsSize);
    // Populate adjacency list structure
    #if !VOXEL
        buildGraphKernelN2<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, neighborLists, listStart, labels, f1, f2);
        checkStatus(hipGetLastError());
        checkStatus(hipDeviceSynchronize());
    #endif
    #if VOXEL
        buildGraphKernel<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, tolerance, neighborLists, listStart, labels, f1, f2, bins);
        checkStatus(hipGetLastError());
        checkStatus(hipDeviceSynchronize());
    #endif

    std::cerr<<"Graph kernel built\n";
    checkStatus(hipGetLastError());
    checkStatus(hipDeviceSynchronize());
    bool stillGoingCPU = true;    
    while(stillGoingCPU) {
        //one iteration of label propogation
        stillGoingCPU = false;
        hipMemcpy(stillGoing, &stillGoingCPU, sizeof(bool), hipMemcpyHostToDevice);
        propogateLabels<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, neighborLists, listStart, labels, f1, f2, stillGoing);

        //swap the frontiers
        bool* t = f1;
        f1 = f2;
        f2 = t;

        //get flag to see if we are done
        hipMemcpy(&stillGoingCPU, stillGoing, sizeof(bool), hipMemcpyDeviceToHost);
    }

    //Build useful data structures mapping points to clusters and clusters to number of points
    //Let C be the number of clusters, and N the number of points in the cloud
    //After we preform the operations in this block, the contents of the vectors are as follows:
    thrust::device_vector<int> labelsSorted(pc.size); //Point labels sorted by cluster. Len(N). 
    thrust::device_vector<int> count(pc.size, 1); //buffer of all 1s. Len(N)
    thrust::device_vector<int> keys(pc.size); //Each clusters unique ID in ascending order Len(C)
    thrust::device_vector<int> values(pc.size); //The number of points in each cluster in ascending order by ID. Len(C)
    thrust::copy(thrust::device, labels, labels+pc.size, labelsSorted.begin()); //first make the labels sorted contain the labels in order of points
    thrust::sort(thrust::device, labelsSorted.begin(), labelsSorted.end()); //now sort the labels by their label idx, 
    auto pair = thrust::reduce_by_key(thrust::device, labelsSorted.begin(), labelsSorted.end(), count.begin(), keys.begin(), values.begin()); //remove duplicate labels and determine the number of points belonging to each label    
   
    //Determine how many clusters there actually are
    
    int numClustersOrig = thrust::distance(keys.begin(), pair.first);

    

    float *minX, *maxX, *minY, *maxY, *minZ, *maxZ; 
    hipMalloc(&minX, sizeof(float)*numClustersOrig);
    hipMalloc(&maxX, sizeof(float)*numClustersOrig);
    hipMalloc(&minY, sizeof(float)*numClustersOrig);
    hipMalloc(&maxY, sizeof(float)*numClustersOrig);
    hipMalloc(&minZ, sizeof(float)*numClustersOrig);
    hipMalloc(&maxZ, sizeof(float)*numClustersOrig);
    thrust::fill(thrust::device, minX, minX + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxX, maxX + numClustersOrig, -std::numeric_limits<float>::max());
    thrust::fill(thrust::device, minY, minY + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxY, maxY + numClustersOrig, -std::numeric_limits<float>::max());
    thrust::fill(thrust::device, minZ, minZ + numClustersOrig, std::numeric_limits<float>::max());
    thrust::fill(thrust::device, maxZ, maxZ + numClustersOrig, -std::numeric_limits<float>::max());

    /*
    //Now get a list of cluster ID keys that are bigger than the min size by removing those that are less than the min size
    is_smaller_than_min pred(minSize);
    auto keyEnd = thrust::remove_if(thrust::device, keys.begin(), keys.end(), values.begin(), pred);
    thrust::remove_if(thrust::device, values.begin(), values.end(), pred);

    int numClusters = keyEnd - keys.begin();
    keys.resize(numClusters);
    values.resize(numClusters);
    std::cout << "CLUSTERS NEW: " << numClusters << std::endl; */

    //find interest points
    //exculsive scan on values to give the indicies of each new cluster start in the points array 
    //for each on the array returned by the exclusive scan, going from the prev element to the cur,
    //first determine if the labels for that range are contained within the clusterIDs [keys] vector (binary search),
    //if so, then find extrema, otherwise move on

    //Call a kernel to color the clusters for debug reasons
    int* gpuKeys = thrust::raw_pointer_cast( keys.data() );
    int* gpuVals = thrust::raw_pointer_cast( values.data() );
    colorClusters<<<ceilDiv(pc.size, MAX_THREADS), MAX_THREADS>>>(pc, labels, gpuKeys, gpuVals, minSize, numClustersOrig, minX, maxX, minY, maxY, minZ, maxZ);

    int * validClustersCount;
    hipMalloc(&validClustersCount, sizeof(int));
    hipMemset(validClustersCount, 0, sizeof(int));
    //colorExtrema<<<ceilDiv(numClustersOrig, MAX_THREADS), MAX_THREADS >>>(pc, gpuVals, minSize, labels, numClustersOrig, validClustersCount, minX, maxX, minY, maxY, minZ, maxZ);
    // TODO: make maxSize do something
    float *minXCPU, *maxXCPU, *minYCPU, *maxYCPU, *minZCPU, *maxZCPU; 
    minXCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxXCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    minYCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxYCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    minZCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    maxZCPU = (float*) malloc(sizeof(float)*numClustersOrig);
    hipMemcpy(minXCPU, minX, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxXCPU, maxX, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(minYCPU, minY, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxYCPU, maxY, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(minZCPU, minZ, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
    hipMemcpy(maxZCPU, maxZ, sizeof(float)*numClustersOrig, hipMemcpyDeviceToHost);
/* 
    int* leftBearing;
    int* rightBearing;
    int* leftCPU;
    int* rightCPU; 

    leftCPU = (int*) malloc(sizeof(int));
    rightCPU = (int*) malloc(sizeof(int));

    hipMalloc(&leftBearing, sizeof(float));
    hipMalloc(&rightBearing, sizeof(float));
    
    //Laucnh kernels to find clear paths using mins and max cluster arrasy
    findClearPathKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, leftBearing, rightBearing);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, leftBearing, 0);
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();

    findAngleOffCenterKernel<<<1, MAX_THREADS>>>(minX, maxX, minZ, maxZ, numClustersOrig, rightBearing, 1);    
    checkStatus(hipGetLastError());
    hipDeviceSynchronize();
    
    //Copy bearings to CPU and display the bearings
    hipMemcpy(leftCPU, leftBearing, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(rightCPU, rightBearing, sizeof(int), hipMemcpyDeviceToHost);
    
    //Set CPU bearings for viewer use
    bearingRight = *rightCPU;
    bearingLeft = *leftCPU;
    
    hipFree(leftBearing);
    hipFree(rightBearing);
    free(leftCPU);
    free(rightCPU);
     */
    checkStatus(hipDeviceSynchronize()); //not needed?
    hipFree(neighborLists);
    hipFree(minX);
    hipFree(maxX);
    hipFree(minY);
    hipFree(maxY);
    hipFree(minZ);
    hipFree(maxZ);

    int validClustersCPU;
    hipMemcpy(&validClustersCPU, validClustersCount, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "valid cluster size: " << validClustersCPU << std::endl;

    ObsReturn obsReturn;
    obsReturn.size = numClustersOrig;
    //copy over elements so we can free the memory later
    Obstacle add;
    for (size_t i = 0; i < numClustersOrig; ++i) {
        //build an obstacle to add to the vector
        add.minX = minXCPU[i];
        add.maxX = maxXCPU[i];
        add.minY = minYCPU[i];
        add.maxY = maxYCPU[i];
        add.minZ = minZCPU[i];
        add.maxZ = maxZCPU[i];

        obsReturn.obs.push_back(add);
    }

    //free memory
    free(minXCPU);
    free(maxXCPU);
    free(minYCPU);
    free(maxYCPU);
    free(minZCPU);
    free(maxZCPU);
    /*
    * obsReturn has a vector of obstacles(named obs),
    which have the min and max of each axis as data members
    */
    return obsReturn;
}
