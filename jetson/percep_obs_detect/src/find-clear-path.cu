#include "hip/hip_runtime.h"
#include "find-clear-path.hpp"
#include <iostream>

//Default findClear Ctor
__device__ FindClearPath::FindClearPath(){}

//Default bearingLine Ctor
__device__ BearingLines::BearingLines() {
  heading = 0;
  n.x = -1;
  n.y = 0;
  bLeft.x = -1;
  bLeft.y = 0;
  bRight.x = 1;
  bRight.y = 0;
}


//Ctor with specified heading
__device__ BearingLines::BearingLines(float heading_in) : heading{heading_in} {
  //NB: Defines heading = 0 as straight, heading > 0 right, heading < 0 left
  n.x = -cos(heading_in); //Calculate x component of orthogonal vec from heading_in
  n.y = sin(heading_in); //Calculate y component of orthogonal vec from heading_in
  bLeft.x = (-rovWidth/2) * cos(heading_in); //Calculate bLeft x offset from heading_in //POSSIBLE ISSUE, LEFT IS POS x TODO: CHECK
  bLeft.y = (rovWidth/2) * sin(heading_in); //Calculate bLeft y offset from heading_in //POSSIBLE ISSUE, LEFT IS POS x
  bRight.x = (rovWidth/2) * cos(heading_in); //Calculate bRight x offset from heading_in //POSSIBLE ISSUE, LEFT IS POS x
  bRight.y = (-rovWidth/2) * sin(heading_in); //Calculate bRight y offset from heading_in //POSSIBLE ISSUE, LEFT IS POS x
}

float2 FindClearPath::find_clear_path_initiate(EuclideanClusterExtractor::ObsReturn obsVec){

  std::cout << "size of obsVec: " << obsVec.obs.size() << std::endl;

  //Allocate and copy obstacle structs array
  EuclideanClusterExtractor::Obstacle* gpuObstacles; 
  hipMalloc(&gpuObstacles, obsVec.obs.size()*sizeof(EuclideanClusterExtractor::Obstacle));
  hipMemcpy(gpuObstacles, &obsVec.obs[0], obsVec.obs.size()*sizeof(EuclideanClusterExtractor::Obstacle), hipMemcpyHostToDevice);
  
  //Allocate heading checks array
  bool* heading_checks;
  hipMalloc(&heading_checks, bearingNum*sizeof(bool));

  double min_distance = 0; 
  //Run find_clear_path on each of the 1024 headings (threads)
  find_clear_path<<<1, bearingNum>>>(gpuObstacles, heading_checks, min_distance, obsVec.obs.size());

  checkStatus(hipDeviceSynchronize());

  //TODO: what to do with heading_checks array
  bool* cpu_heading_checks = new bool[bearingNum];
  hipMemcpy(cpu_heading_checks, heading_checks, bearingNum, hipMemcpyDeviceToHost);

  // Prints out heading_check array
  // for(int i = 0; i < bearingNum; ++i){
  //   std::cout << cpu_heading_checks[i] << " ";
  // }
  // std::cout << std::endl;

  //Find closest heading to the left and right of our current heading
  int heading_left = find_left_closest(cpu_heading_checks);
  int heading_right = find_right_closest(cpu_heading_checks);

  //TODO cout in the driver
  // std::cout << "left heading: " << heading_left << std::endl;
  // std::cout << "right heading: " << heading_right << std::endl;

  //Free memory
  hipFree(gpuObstacles);
  hipFree(heading_checks);

  float3 output;
  output.x = heading_left;
  output.y = heading_right;
  output.min_dist = min_distance; 
  return output;
}

__global__ void find_clear_path(EuclideanClusterExtractor::Obstacle* obstacles, bool* heading_checks, double &min_distance, int obsArrSize){
  
  int i = threadIdx.x;
  heading_checks[i] = 1; //Assume a clear heading
 
  int map = (i - bearingNum/2);

  float bearing_deg = float(map * fov) / (bearingNum / 2); //converts thread # to degrees //TODO Bring this back

  BearingLines bearings(bearing_deg * 3.1415926535/180.0); //Create bearing lines from bearing //TODO how accurate should pi be?

  min_distance = std::numeric_limits<double>::max();         
  // if detect variables are negative, obs is to the right of bearing line
  // if detect variables are positive, obs is to the left of bearing line
  // if detect variables are 0, obs is on the bearing line
  for(int j = 0; j < obsArrSize; ++j){ //Check all obstacles in obstacles array
      if(obstacles[j].minX < obstacles[j].maxX && obstacles[j].minZ < obstacles[j].maxZ){ 

        //Note: botLeft = minX, minZ    botRight = maxX, minZ    topLeft = minX, maxZ    topRight = maxX, maxZ
        float LBL_botLeft = (bearings.n.x * (obstacles[j].minX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].minZ - bearings.bLeft.y));
        float RBL_botLeft = (bearings.n.x * (obstacles[j].minX - bearings.bRight.x)) + (bearings.n.y * (obstacles[j].minZ - bearings.bRight.y));

        float LBL_botRight = (bearings.n.x * (obstacles[j].maxX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].minZ - bearings.bLeft.y));
        float RBL_botRight = (bearings.n.x * (obstacles[j].maxX - bearings.bRight.x)) + (bearings.n.y * (obstacles[j].minZ - bearings.bRight.y));

        float LBL_topLeft = (bearings.n.x * (obstacles[j].minX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].maxZ - bearings.bLeft.y));
        float RBL_topLeft = (bearings.n.x * (obstacles[j].minX - bearings.bRight.x)) + (bearings.n.y * (obstacles[j].maxZ - bearings.bRight.y));

        float LBL_topRight = (bearings.n.x * (obstacles[j].maxX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].maxZ - bearings.bLeft.y));
        float RBL_topRight = (bearings.n.x * (obstacles[j].maxX - bearings.bRight.x)) + (bearings.n.y * (obstacles[j].maxZ - bearings.bRight.y));

        // Check if obstacle its between bearing lines
        if((LBL_botLeft > 0 && RBL_botLeft < 0) || (LBL_botLeft < 0 && RBL_botLeft > 0)
            || LBL_botLeft == 0 || RBL_botLeft == 0){ 
          heading_checks[i] = 0; // This is not a clear path
        }

        if((LBL_botRight > 0 && RBL_botRight < 0) || (LBL_botRight < 0 && RBL_botRight > 0)
            || LBL_botRight == 0 || RBL_botRight == 0){ 
          heading_checks[i] = 0; // This is not a clear path
        }

        if((LBL_topLeft > 0 && RBL_topLeft < 0) || (LBL_topLeft < 0 && RBL_topLeft > 0)
            || LBL_topLeft == 0 || RBL_topLeft == 0){ 
          heading_checks[i] = 0; // This is not a clear path
        }

        if((LBL_topRight > 0 && RBL_topRight < 0) || (LBL_topRight < 0 && RBL_topRight > 0)
            || LBL_topRight == 0 || RBL_topRight == 0){ 
          heading_checks[i] = 0; // This is not a clear path
        }

        // Check if obstacle is larger than span of bearing lines
        if((LBL_botLeft > 0 && RBL_botRight < 0) || (LBL_topLeft > 0 && RBL_topRight < 0))
        {
          heading_checks[i] = 0; // This is not a clear path
        }

        //MIN DISTANCE
        if(i == 0) //run once per obstacle
        {
          double dist = std::sqrt(std::min(pow(obstacles[i].minX, 2), pow(obstacles[i].maxX, 2)) 
                        + std::min(pow(obstacles[i].minZ, 2), pow(obstacles[i].maxZ, 2)));
          if(dist < min_distance)
          {
            min_distance = dist; 
          }
        } 
      } //end valid obstacle check (if)
  } // end loop 
}

//Find first clear bearing to the left of our straight ahead bearing and convert it to a degree bearing
float FindClearPath::find_left_closest(bool* headings){
  int idx = bearingNum/2; //Start at 0 heading 
  int clear = 0;
  //Find first clear heading in the heading array and return it
  while(clear == 0 && idx >= 0){ 
    if(headings[idx] == 1){
      clear = 1;
    }
    --idx;
  }
  return (fov*((idx+1)-(bearingNum/2)))/(bearingNum/2);
}
    
//Find first clear bearing to the right of our straight ahead bearing and convert it to a degree bearing
float FindClearPath::find_right_closest(bool* headings){
  int idx = bearingNum/2; //Start at 0 heading
  int clear = 0;
  //Find first clear heading in the heading array and return it
  while(clear == 0 && idx < bearingNum){ 
    if(headings[idx] == 1){
      clear = 1;
    }
    ++idx;
  }
  return (fov*((idx+1)-(bearingNum/2)))/(bearingNum/2);
}