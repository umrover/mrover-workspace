#include "hip/hip_runtime.h"
#include "find-clear-path.hpp"
#include <iostream>

//Default findClear Ctor
__device__ FindClearPath::FindClearPath(){}

//Default bearingLine Ctor
__device__ BearingLines::BearingLines() {
  heading = 0;
  n.x = -1;
  n.y = 0;
  bLeft.x = -1;
  bLeft.y = 0;
  bRight.x = 1;
  bRight.y = 0;
}


//Ctor with specified heading
__device__ BearingLines::BearingLines(float heading_in) : heading{heading_in} {
  //NB: Defines heading = 0 as straight, heading > 0 right, heading < 0 left
  n.x = -cos(heading_in); //Calculate x component of orthogonal vec from heading_in
  n.y = sin(heading_in); //Calculate y component of orthogonal vec from heading_in
  bLeft.x = (-rovWidth/2) * cos(heading_in); //Calculate bLeft x offset from heading_in
  bLeft.y = (rovWidth/2) * sin(heading_in); //Calculate bLeft y offset from heading_in
  bRight.x = (rovWidth/2) * cos(heading_in); //Calculate bRight x offset from heading_in
  bRight.y = (-rovWidth/2) * sin(heading_in); //Calculate bRight y offset from heading_in
}

void FindClearPath::find_clear_path_initiate(EuclideanClusterExtractor::ObsReturn obsVec){
  //Allocate and copy obstacle structs array
  EuclideanClusterExtractor::Obstacle* gpuObstacles; 
  hipMalloc(&gpuObstacles, obsVec.obs.size()*sizeof(EuclideanClusterExtractor::Obstacle));
  hipMemcpy(gpuObstacles, &obsVec.obs[0], obsVec.obs.size()*sizeof(EuclideanClusterExtractor::Obstacle), hipMemcpyHostToDevice);
  
  //Allocate heading checks array
  bool* heading_checks;
  hipMalloc(&heading_checks, bearingNum*sizeof(bool));

  //Run find_clear_path on each of the 1024 headings (threads)
  find_clear_path<<<1, bearingNum>>>(gpuObstacles, heading_checks, obsVec.obs.size());

  checkStatus(hipDeviceSynchronize());

  //TODO: what to do with heading_checks array
  bool* cpu_heading_checks = new bool[bearingNum];
  hipMemcpy(cpu_heading_checks, heading_checks, bearingNum, hipMemcpyDeviceToHost);

  //Find closest heading to the left and right of our current heading
  int heading_left = find_left_closest(cpu_heading_checks);
  int heading_right = find_right_closest(cpu_heading_checks);

  //TODO cout in the driver
  std::cout << "left heading: " << heading_left << std::endl;
  std::cout << "right heading: " << heading_right << std::endl;

  //Free memory
  hipFree(gpuObstacles);
  hipFree(heading_checks);
}

__global__ void find_clear_path(EuclideanClusterExtractor::Obstacle* obstacles, bool* heading_checks, int obsArrSize){
  
  int i = threadIdx.x;
  heading_checks[i] = 1; //Assume a clear heading

  //Create bearing lines based on threadIdx
  //NB: threadIdx 511 is the 0 heading

  //fov: one directional field of view, currently 80 degrees
  //bearingNum: number of gpu threads
  //i is index of thread
  int bearing = int(i - bearingNum/2);
  bearing = (bearing * fov) / (bearingNum / 2); //converts thread # to degrees
  BearingLines bearings(bearing); //Create bearing lines from bearing

  //TODO: FIX LOGIC
  // if detect variables are negative, obs is to the left of bearing line
  // if detect variables are positive, obs is to the right of bearing line
  // if detect variables are 0, obs is on the bearing line
  for(int j = 0; j < obsArrSize; ++j){ //Check all obstacles in obstacles array
    //checks the left bearing at the minimum y of the object
    float detectLmin = (bearings.n.x * (obstacles[j].minX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].minY - bearings.bLeft.y));

    //checks the left bearing at the maximum y of the object
    float detectLmax = (bearings.n.x * (obstacles[j].maxX - bearings.bLeft.x)) + (bearings.n.y * (obstacles[j].maxY - bearings.bLeft.y));

    //checks the right bearing at the minimum y of the object
    float detectRmin = (bearings.n.x * (obstacles[j].minX - bearings.bRight.y)) + (bearings.n.y * (obstacles[j].minY - bearings.bRight.y));

    //check the right bearing at the maximum y of the object
    float detectRmax = (bearings.n.x * (obstacles[j].maxX - bearings.bRight.y)) + (bearings.n.y * (obstacles[j].maxY - bearings.bRight.y));

    bool min_test = (detectLmin < 0 && detectRmin < 0) || (detectLmin > 0 && detectRmin > 0); //checks that the left bearing and the right bearing are on the same side for 
    bool max_test = (detectLmax < 0 && detectRmax < 0) || (detectLmax > 0 && detectRmax > 0);
    if(min_test && max_test){ //If to the left of left bearing and right of right bearing
      heading_checks[i] = 0; //This is not a clear heading
    }
  }
}

//Find first clear bearing to the left of our straight ahead bearing and convert it to a degree bearing
int FindClearPath::find_left_closest(bool* headings){
  int idx = bearingNum/2; //Start at 0 heading 
  int clear = 0;
  //Find first clear heading in the heading array and return it
  for(int i = bearingNum/2; i >= 0; --i){ 
    if(headings[i] == 1){
      idx == i;
      break;
    }
  }
  return (int)(fov*(idx-(int)(bearingNum/2)))/(int)(bearingNum/2);
}
    
//Find first clear bearing to the right of our straight ahead bearing and convert it to a degree bearing
int FindClearPath::find_right_closest(bool* headings){
  int idx = bearingNum/2; //Start at 0 heading
  int clear = 0;
  //Find first clear heading in the heading array and return it
  for(int i = bearingNum/2; i<bearingNum; ++i){
    if(headings[i] == 1){
      idx == i;
      break;
    }
  }
  return (int)(fov*(idx-(int)(bearingNum/2)))/(int)(bearingNum/2);
}